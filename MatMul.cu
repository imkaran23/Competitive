#include "hip/hip_runtime.h"
// Karan - 18je0395

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#define Tile_size 2

//Function To handle any errors occurred in the function calls
#define funcCheck(stmt) do {
        hipError_t err = stmt;
        if (err != hipSuccess) {
            printf( "Failed to run stmt %d ", __LINE__);
            return -1;
        }
    } while(0)

int numARows;   // number of rows in the matrix A
int numAColumns;  // number of columns in the matrix A
int numBRows;   // number of rows in the matrix B
int numBColumns;  // number of columns in the matrix B
int numCRows;  // number of rows in the matrix C (you have to set this)
int numCColumns; // number of columns in the matrix C (you have to set this)

// Compute C = A * B by Kernel for shared memory/ Tiled execution
__global__ void matrixMultiplyShared(float * A, float * B, float * C,
                                    int numARows, int numAColumns,
                                    int numBRows, int numBColumns,
                                    int numCRows, int numCColumns)
{
    __shared__ float sA[Tile_size][Tile_size];   // Tile size to store elements in shared memory
    __shared__ float sB[Tile_size][Tile_size];

    int Row = blockDim.y*blockIdx.y + threadIdx.y; //To generate ids of threads.
    int Col = blockDim.x*blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int k = 0; k < (((numAColumns - 1)/ Tile_size) + 1); k++)
    {
        if ( (Row < numARows) && (threadIdx.x + (k*Tile_size)) < numAColumns)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sA[threadIdx.y][threadIdx.x] = A[(Row*numAColumns) + threadIdx.x + (k*Tile_size)];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }
        if ( Col < numBColumns && (threadIdx.y + k*Tile_size) < numBRows)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*Tile_size)*numBColumns + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < Tile_size; ++j)//Multiplying Elements present in tile
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    if (Row < numCRows && Col < numCColumns)//Saving Final result into Matrix C
    {
        C[Row*numCColumns + Col] = Cvalue;
    }
}

void Print_Mat(int Row,int Col,float * Mat)//Function To print the Matrix
{
 for(int i=0;i<Row*Col;i++)
   {
   printf("%f  ",*(Mat+i));

   if((i%Col)==0 )
    {
     printf("\n");
    }
   }
}

int main(int argc, char ** argv) {
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * hostComputedC;
    float * deviceA;
    float * deviceB;
    float * deviceC;

    printf("\nPlease Enter Rows and Columns of A:");
    scanf("%d %d",&numARows,&numAColumns);

    printf("\nPlease Enter Rows and Columns of B:");
    scanf("%d %d",&numBRows,&numBColumns);

    hostA = (float *) malloc(sizeof(float)*numARows*numAColumns);
    hostB = (float *) malloc(sizeof(float)*numBRows*numBColumns);

    for (int i = 0; i < numARows*numAColumns; i++)//Matrix Initialization
    {
        hostA[i]=1.0;
    }
    for (int i = 0; i < numBRows*numBColumns; i++)
    {
        hostB[i]=1.0;
    }

    printf("\nMatrix A Values:\n");
    Print_Mat(numARows,numAColumns,hostA);//Function Call

    printf("\n\nMatrix B Values:\n");
    Print_Mat(numBRows,numBColumns,hostB);//Function Call

    numCRows = numARows;
    numCColumns = numBColumns;

    hostC = (float *) malloc(sizeof(float)*numCRows*numCColumns);
    hostComputedC = (float *) malloc(sizeof(float)*numCRows*numCColumns);

    // Allocating GPU memory
    funcCheck(hipMalloc((void **)&deviceA, sizeof(float)*numARows*numAColumns));
    funcCheck(hipMalloc((void **)&deviceB, sizeof(float)*numBRows*numBColumns));
    funcCheck(hipMalloc((void **)&deviceC, sizeof(float)*numCRows*numCColumns));

    // Copy memory to the GPU
    funcCheck(hipMemcpy(deviceA, hostA, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice));
    funcCheck(hipMemcpy(deviceB, hostB, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice));

    // Initialize the grid and block dimensions
    dim3 dimGrid((numCColumns/Tile_size) + 1, (numCRows/Tile_size) + 1, 1);//Number of Blocks required
    dim3 dimBlock(Tile_size, Tile_size, 1);//Number of threads in each block

    //@@ Launch the GPU Kernel here
    matrixMultiplyShared<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

    hipDeviceSynchronize();//To synchronize the device

    // Copy the results in GPU memory back to the CPU
    funcCheck(hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns, hipMemcpyDeviceToHost));

    printf("\nMatrix C From Device\n");
    Print_Mat(numCRows,numCColumns,hostC);//Function Call

    printf("\n Number of Blocks Created:%d \n",((numCColumns/Tile_size) + 1)*((numCColumns/Tile_size) + 1));
    printf("\n Number of Threads Per Block: %d \n",(Tile_size*Tile_size));

    // Free the GPU memory
    funcCheck(hipFree(deviceA));
    funcCheck(hipFree(deviceB));
    funcCheck(hipFree(deviceC));
    //Free the Pointer Memory
    free(hostA);
    free(hostB);
    free(hostC);
    free(hostComputedC);

    return 0;
}