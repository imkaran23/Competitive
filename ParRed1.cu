#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 512 
#define NUM_OF_ELEMS 4096 

#define funcCheck(stmt) {                                            \
    hipError_t err = stmt;                                          \
    if (err != hipSuccess)                                          \
    {                                                                \
        printf( "Failed to run stmt %d ", __LINE__);                 \
        printf( "Got CUDA error ...  %s ", hipGetErrorString(err)); \
        return -1;                                                   \
    }                                                                \
}

__global__  void total(float * input, float * output, int len) 
{
    // Load a segment of the input vector into shared memory
    __shared__ float partialSum[2*BLOCK_SIZE];
    int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int t = threadIdx.x;
    unsigned int start = 2*blockIdx.x*blockDim.x;

    if ((start + t) < len)
    {
        partialSum[t] = input[start + t];      
    }
    else
    {       
        partialSum[t] = 0.0;
    }
    if ((start + blockDim.x + t) < len)
    {   
        partialSum[blockDim.x + t] = input[start + blockDim.x + t];
    }
    else
    {
        partialSum[blockDim.x + t] = 0.0;
    }

    // Traverse reduction tree
    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
    {
      __syncthreads();
        if (t < stride)
            partialSum[t] += partialSum[t + stride];
    }
    __syncthreads();

    // Write the computed sum of the block to the output vector at correct index
    if (t == 0 && (globalThreadId*2) < len)
    {
        output[blockIdx.x] = partialSum[t];
    }
}

int main(int argc, char ** argv) 
{
    int ii;

    float * hostInput; // The input 1D vector
    float * hostOutput; // The output vector
    float * deviceInput;
    float * deviceOutput;

    int numInputElements = NUM_OF_ELEMS; // number of elements in the input list
    int numOutputElements; // number of elements in the output list
    hostInput = (float *) malloc(sizeof(float) * numInputElements);

    for (int i=0; i < NUM_OF_ELEMS; i++)
    {
        hostInput[i] = 1.0;     // Add your input values
    }

    numOutputElements = numInputElements / (BLOCK_SIZE<<1);
    if (numInputElements % (BLOCK_SIZE<<1)) 
    {
        numOutputElements++;
    }
    hostOutput = (float*) malloc(numOutputElements * sizeof(float));

    funcCheck(hipMalloc((void **)&deviceInput, numInputElements * sizeof(float)));
    funcCheck(hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float)));

    hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);

    dim3 DimGrid( numOutputElements, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    total<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
    hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);

    // Reduce output vector on the host
    for (ii = 1; ii < numOutputElements; ii++) 
    {
        hostOutput[0] += hostOutput[ii];
    }

    printf("Reduced Sum from GPU = %f\n", hostOutput[0]);   

    hipFree(deviceInput);
    hipFree(deviceOutput); 
    free(hostInput);
    free(hostOutput);

    return 0;
}